#include "hip/hip_runtime.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include "transpose_device.cuh"

/*
 * TODO for all kernels (including naive):
 * Leave a comment above all non-coalesced memory accesses and bank conflicts.
 * Make it clear if the suboptimal access is a read or write. If an access is
 * non-coalesced, specify how many cache lines it touches, and if an access
 * causes bank conflicts, say if its a 2-way bank conflict, 4-way bank
 * conflict, etc.
 *
 * Comment all of your kernels.
 */


/*
 * Each block of the naive transpose handles a 64x64 block of the input matrix,
 * with each thread of the block handling a 1x4 section and each warp handling
 * a 32x4 section.
 *
 * If we split the 64x64 matrix into 32 blocks of shape (32, 4), then we have
 * a block matrix of shape (2 blocks, 16 blocks).
 * Warp 0 handles block (0, 0), warp 1 handles (1, 0), warp 2 handles (0, 1),
 * warp n handles (n % 2, n / 2).
 *
 * This kernel is launched with block shape (64, 16) and grid shape
 * (n / 64, n / 64) where n is the size of the square matrix.
 *
 * You may notice that we suggested in lecture that threads should be able to
 * handle an arbitrary number of elements and that this kernel handles exactly
 * 4 elements per thread. This is OK here because to overwhelm this kernel
 * it would take a 4194304 x 4194304    matrix, which would take ~17.6TB of
 * memory (well beyond what I expect GPUs to have in the next few years).
 */
__global__
void naiveTransposeKernel(const float *input, float *output, int n) {
    const int i = threadIdx.x + 64 * blockIdx.x;
    int j = 4 * threadIdx.y + 64 * blockIdx.y;
    const int end_j = j + 4;

    for (; j < end_j; j++)
        output[j + n * i] = input[i + n * j];
    /* Since the naive kernel doesn't use shared memory, we do not have
     * bank conflict. The remaining problem is data alignment.
     * Each warp handles 32*4 = 128 4-bytes elements, hence there is minimum
     * of 4 cache reads. However, in here, due to the fact that n >= 512,
     * each thread in a warp reads from 5 cache lines. A warp reads 160 cache
     * lines.
     */
}

__global__
void shmemTransposeKernel(const float *input, float *output, int n) {

    __shared__ float data[64*64];
    int s_i = threadIdx.x;
    int s_j = threadIdx.y * 4;
    int s_idx = s_j + s_i * 16;

    const int i = threadIdx.x + 64 * blockIdx.x;
    int j = 4 * threadIdx.y + 64 * blockIdx.y;
    const int end_j = j + 4;

    for (int jj = j; jj < end_j; jj++)
        data[s_i++] = input[i + n * jj];
    __syncthreads();

    for (int jj = j; jj < end_j; jj++)
        output[jj + n * i] = data[--s_i];
}

__global__
void optimalTransposeKernel(const float *input, float *output, int n) {
    // TODO: This should be based off of your shmemTransposeKernel.
    // Use any optimization tricks discussed so far to improve performance.
    // Consider ILP and loop unrolling.

    const int i = threadIdx.x + 64 * blockIdx.x;
    int j = 4 * threadIdx.y + 64 * blockIdx.y;
    const int end_j = j + 4;

    for (; j < end_j; j++)
        output[j + n * i] = input[i + n * j];
}

void cudaTranspose(
    const float *d_input,
    float *d_output,
    int n,
    TransposeImplementation type)
{
    if (type == NAIVE) {
        dim3 blockSize(64, 16);
        dim3 gridSize(n / 64, n / 64);
        naiveTransposeKernel<<<gridSize, blockSize>>>(d_input, d_output, n);
    }
    else if (type == SHMEM) {
        dim3 blockSize(64, 16);
        dim3 gridSize(n / 64, n / 64);
        shmemTransposeKernel<<<gridSize, blockSize>>>(d_input, d_output, n);
    }
    else if (type == OPTIMAL) {
        dim3 blockSize(64, 16);
        dim3 gridSize(n / 64, n / 64);
        optimalTransposeKernel<<<gridSize, blockSize>>>(d_input, d_output, n);
    }
    // Unknown type
    else
        assert(false);
}
