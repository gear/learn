#include "hip/hip_runtime.h"
/* 
 * CUDA blur
 * Kevin Yuh, 2014 
 * Revised by Nailen Matschke, 2016
 */

#include <cstdio>

#include <hip/hip_runtime.h>

#include "blur_device.cuh"


__global__
void cudaBlurKernel(const float *raw_data, const float *blur_v, float *out_data,
    int n_frames, int blur_v_size) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < blur_v_size - 1) {

	}
}


void cudaCallBlurKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        const float *raw_data,
        const float *blur_v,
        float *out_data,
        const unsigned int n_frames,
        const unsigned int blur_v_size) {
        
    // TODO: Call the kernel above this function.
}
